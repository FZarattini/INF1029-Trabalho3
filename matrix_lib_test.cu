#include <stdio.h>
#include "matrix_lib.h"

struct matrix matrixA, matrixB, matrixC;

int load_matrix(struct matrix *matrix, char *filename) {
        unsigned long int i = 0;
        unsigned long int N = 0;
        FILE *fd = NULL;

        /* Check the numbers of the elements of the matrix */
        N = matrix->height * matrix->width;

        /* Check the integrity of the matrix */
        if (N == 0 || matrix->rows == NULL) return 0;

        /* Try to open file of floats */
        if ((fd = fopen(filename, "rb")) == NULL) {
            printf("Unable to open file %s\n", filename);
            return 0;
        }

        float *nxt_a = matrix->h_rows; 

        for ( i = 0;
	        i < N; 
	        i += 8, nxt_a += 8) {

	        if (fread(nxt_a, sizeof(float), 8, fd) != 8) {
                printf("Error reading from file %s: short read (less than 8 floats)\n", filename);
                return 0;
	        }
        }

        if (fd != NULL) fclose(fd);

        return 1;
}

int main(void){
    unsigned long int DimA_M, DimA_N, DimB_M, DimB_N;
    char *matrixA_filename, *matrixB_filename, *result1_filename, *result2_filename;
    char *eptr = NULL;
    hipError_t hipError_t;

    // Check arguments
    if (argc != 10) {
            printf("Usage: %s <scalar_value> <DimA_M> <DimA_N> <DimB_M> <DimB_N> <matrixA_filename> <matrixB_filename> <result1_filename> <result2_filename>\n", argv[0]);
            return 0;
    } else {
            //printf("Number of args: %d\n", argc);
            //for (int i=0; i<argc; ++i)
            //       printf("argv[%d] = %s\n", i, argv[i]);
    }

    // Convert arguments
    scalar_value = strtof(argv[1], NULL);
    DimA_M = strtol(argv[2], &eptr, 10);
    DimA_N = strtol(argv[3], &eptr, 10);
    DimB_M = strtol(argv[4], &eptr, 10);
    DimB_N = strtol(argv[5], &eptr, 10);
    matrixA_filename = argv[6];
    matrixB_filename = argv[7];
    result1_filename = argv[8];
    result2_filename = argv[9];

    if ((scalar_value == 0.0f) || (DimA_M == 0) || (DimA_N == 0) || (DimB_M == 0) || (DimB_N == 0)) {
            printf("%s: erro na conversao do argumento: errno = %d\n", argv[0], errno);

            /* If a conversion error occurred, display a message and exit */
            if (errno == EINVAL)
            {
                printf("Conversion error occurred: %d\n", errno);
                return 1;
            }

            /* If the value provided was out of range, display a warning message */
            if (errno == ERANGE) {
                printf("The value provided was out of rangei: %d\n", errno);
                return 1;
        }
    }

    /* Allocate the arrays of the four matrixes */
    float *a=  (float*)aligned_alloc(32, DimA_M*DimA_N*sizeof(float));
    float *b = (float*)aligned_alloc(32, DimB_M*DimB_N*sizeof(float));
    float *c = (float*)aligned_alloc(32, DimA_M*DimB_N*sizeof(float));

    if ((a == NULL) || (b == NULL) || (c == NULL)) {
        printf("%s: array allocation problem.", argv[0]);
        return 1;
    }

    float *h_a=  (float*)malloc(DimA_M * DimA_N * sizeof(float));
    float *h_b = (float*)malloc(DimB_M * DimB_N * sizeof(float));
    float *h_c = (float*)malloc(DimA_M * DimB_N * sizeof(float));


    float *d_a;
    float *d_b;
    float *d_c;

    hipError_t = hipMalloc(&d_a, DimA_M * DimA_N * sizeof(float));
    hipError_t = hipMalloc(&d_b, DimB_M * DimB_N * sizeof(float));
    hipError_t = hipMalloc(&d_c, DimA_M * DimB_N * sizeof(float));

    matrixA.height = DimA_M;
    matrixA.width = DimA_N;
    matrixA.h_rows = h_a;
    matrixA.d_rows = d_a;

    if (!load_matrix(&matrixA, matrixA_filename)) {
        printf("%s: matrixA initialization problem.", argv[0]);
        return 1;
    }
        
    matrixB.height = DimB_M;
    matrixB.width = DimB_N;
    matrixB.h_rows = h_b;
    matrixB.d_rows = d_b;

    if (!load_matrix(&matrixB, matrixB_filename)) {
        printf("%s: matrixB initialization problem.", argv[0]);
        return 1;
    }

    matrixC.height = DimA_M;
    matrixC.width = DimB_N;
    matrixC.h_rows = h_c;
    matrixC.d_rows = d_c;

    hipError_t = hipMemcpy(d_a, h_a, DimA_M * DimA_N * sizeof(float), hipMemcpyHostToDevice);

    hipError_t = hipMemcpy(d_b, h_b, DimB_M * DimB_N * sizeof(float), hipMemcpyHostToDevice);

    hipError_t = hipMemcpy(d_c, h_c, DimA_M * DimB_N * sizeof(float), hipMemcpyHostToDevice);
    

    return 1;
}